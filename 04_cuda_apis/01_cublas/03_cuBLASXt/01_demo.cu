#include <cublasXt.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>

const int M = 1024 / 4;
const int N = 1024 / 4;
const int K = 1024 / 4;

#define CHECK_CUBLAS(call) { hipblasStatus_t err = call; if (err != HIPBLAS_STATUS_SUCCESS) { std::cerr << "Error in " << #call << ", line " << __LINE__ << std::endl; exit(1); } }

// Observe that there are no cuda memcopy in this code. cuBlasXt handles it for us but at an performance overhead.

// cuBLAS-Xt is designed for high-performance computing and is optimized for large-scale matrix operations. 
// Its architecture is based on a tile-based approach, where matrices are divided into smaller tiles that are processed parallel. 
// This approach allows for efficient use of the GPU's memory hierarchy and enables high-bandwidth data transfer between the GPU and the host system.

int main() {
    // Initialize random number generator
    srand(time(0));

    // Allocate host memory for matrices
    float* A_host = new float[M * K];
    float* B_host = new float[K * N];
    float* C_host_cpu = new float[M * N];
    float* C_host_gpu = new float[M * N];

    // Initialize matrices with random values
    for (int i = 0; i < M * K; i++) {
        A_host[i] = (float)rand() / RAND_MAX;
    }
    for (int i = 0; i < K * N; i++) {
        B_host[i] = (float)rand() / RAND_MAX;
    }

    float alpha = 1.0f;
    float beta = 0.0f;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            C_host_cpu[i * N + j] = 0.0f;
            for (int k = 0; k < K; k++) {
                C_host_cpu[i * N + j] += A_host[i * K + k] * B_host[k * N + j];
            }
        }
    }

    cublasXtHandle_t handle;
    CHECK_CUBLAS(cublasXtCreate(&handle));


    int devices[1] = {0};
    CHECK_CUBLAS(cublasXtDeviceSelect(handle, 1, devices));

    // Warmup run
    CHECK_CUBLAS(cublasXtSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B_host, N, A_host, K, &beta, C_host_gpu, N));

    float max_diff = 1e-4f;
    for (int i = 0; i < M * N; i++) {
        float diff = std::abs(C_host_cpu[i] - C_host_gpu[i]);
        if (diff > max_diff) {
            std::cout << "i: " << i << " CPU: " << C_host_cpu[i] << ", GPU: " << C_host_gpu[i] << std::endl;
            
        }
    }

    std::cout << "Maximum difference between CPU and GPU results: " << max_diff << std::endl;

    // Free memory
    delete[] A_host;
    delete[] B_host;
    delete[] C_host_cpu;
    delete[] C_host_gpu;


    return 0;

}

// Output:
// ~/Git/CudaExploration/04_cuda_apis/01_cublas/03_cuBLASXt$ nvcc -lcublas -o ./01_exec ./01_demo.cu 
// ~/Git/CudaExploration/04_cuda_apis/01_cublas/03_cuBLASXt$ ./01_exec 
// Maximum difference between CPU and GPU results: 0.0001
